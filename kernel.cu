#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include "definitions.h"

// Cuda Header
hipError_t calculateDistanceCuda(Point *pointArray, int sizeOfPointArray,
Distance *distanceArray, int sizeOfDistanceArray,int arrayPointer,int num);

__global__ void calculate(Point *pointArray, int sizeOfPointArray,
Distance *distanceArray, int sizeOfDistanceArray,int block,int arrayPointer,int num)
{
for (int i = 0; i < num ; i++)
{
int myThreadId = threadIdx.x + blockIdx.x * block; //my index in all GPU
float tempx,tempy,res;

float myx = pointArray[i + arrayPointer].x;
float myy = pointArray[i + arrayPointer].y;
res = powf(abs(myx-pointArray[myThreadId].x),2) + powf(abs(myy-pointArray[myThreadId].y),2);
distanceArray[myThreadId+i*sizeOfPointArray].id = myThreadId;
distanceArray[myThreadId+i*sizeOfPointArray].distance = sqrt((float)res);
}
}

// The main method that run in main.ccp
hipError_t calculateDistanceCuda(Point *pointArray, int sizeOfPointArray,
Distance *distanceArray, int sizeOfDistanceArray, int arrayPointer,int num)
{
hipError_t cudaStatus;
Point* dev_pointArray = 0;
Distance*  dev_distanceArray = 0;

cudaStatus = hipMalloc((void**)&dev_pointArray, sizeOfPointArray*sizeof(Point));

cudaStatus = hipMalloc((void**)&dev_distanceArray, sizeOfDistanceArray*sizeof(Distance));

// Copy pointArray array from host memory to GPU buffers.
cudaStatus = hipMemcpy(dev_pointArray,pointArray, sizeOfPointArray * sizeof(Point), hipMemcpyHostToDevice);

int blocks = sizeOfPointArray/NUMOFTHREADS;

// Launch a kernel on the GPU
calculate<<< blocks, NUMOFTHREADS >>>(dev_pointArray, sizeOfPointArray, dev_distanceArray, sizeOfDistanceArray, NUMOFTHREADS,arrayPointer ,num);

cudaStatus = hipDeviceSynchronize();

// Copy dev_distanceArray  from GPU buffer to host memory.
cudaStatus = hipMemcpy(distanceArray, dev_distanceArray, sizeOfDistanceArray*sizeof(Distance), hipMemcpyDeviceToHost);

return cudaStatus;
}
